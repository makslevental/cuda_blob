#include "hip/hip_runtime.h"
//
// Created by Maksim Levental on 12/28/20.
//

#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <vector>

#include "hipfft/hipfftXt.h"
#include "hip/hip_runtime_api.h"
#include "multigpu.cuh"

void print3Dfloat2(std::vector<float2> hOut, int nb, int nr, int nc) {
    for (int k = 0; k < nb; k++) {
        for (int i = 0; i < nr; i++) {
            for (int j = 0; j < nc; j++) {
                auto zzyyxx = (k * (nr * nc)) + (i * nc) + j;
                printf("%.2f + %.2fi    ", hOut[zzyyxx].x, hOut[zzyyxx].y);
            }
            printf("\n");
        }
        printf("\nbatch %d *************\n", k);
    }
}

void print3D(std::vector<float> hOut, int nb, int nr, int nc) {
    for (int k = 0; k < nb; k++) {
        for (int i = 0; i < nr; i++) {
            printf("[ ");
            for (int j = 0; j < nc; j++) {
                printf("%.2f,   ", hOut[k * (nr * nc) + i * nc + j]);
            }
            printf("],\n ");
        }
        printf("\nbatch %d *************\n", k);
    }
}

int runMultiGPU() {
    static const int numGPUs = 2;
    int gpus[numGPUs] = {0, 1};

    // The real-to-complex transform is implicitly a forward transform. For an in-place
    // real-to-complex transform where FFTW compatible output is desired, the input size must be
    // padded to ⌊N/2⌋ + 1 complex elements.
    // wtf?
    int nb = 16;
    int nr = 8;
    int nc = 8;

    // Fill with junk data
    std::vector<float> hIn(nb * nr * nc);
    //    for (int i = 0; i < nb * nr * nc; ++i) {
    //        hIn[i] = static_cast<float>(i);
    //    }
    for (int k = 0; k < nb; k++) {
        for (int i = 0; i < nr; i++) {
            for (int j = 0; j < nc; j++) {
                hIn[k * (nr * nc) + i * nc + j] = i * nc + j;
            }
        }
    }

    //    print3D(hIn, nb, nr, nc);

    hipfftHandle plan;
    checkCudaErrors(hipfftCreate(&plan));
    checkCudaErrors(hipfftXtSetGPUs(plan, numGPUs, gpus));

    // dimension of fft
    int rank = 2;
    int n[2] = {nr, nc};
    // input/output sizes with pitches ("unpitched")
    int inEmbed[] = {nr, nc};
    int onEmbed[] = {nr, nc / 2 + 1};
    // dist between batches
    int iDist = nr * nc;
    int oDist = nr * (nc / 2 + 1);
    // stride between adjacent entries in row
    int iStride = 1;
    int oStride = 1;

    size_t workSize[2];
    hipfftMakePlanMany(
        plan, rank, n, inEmbed, iStride, iDist, onEmbed, oStride, oDist, HIPFFT_R2C, nb, workSize);

    hipLibXtDesc* dX;
    checkCudaErrors(hipfftXtMalloc(plan, &dX, HIPFFT_XT_FORMAT_INPLACE));

    checkCudaErrors(hipfftXtMemcpy(plan, dX, (void*)hIn.data(), HIPFFT_COPY_HOST_TO_DEVICE));

    checkCudaErrors(hipfftXtExecDescriptorR2C(plan, dX, dX));
    checkCudaErrors(hipGetLastError());

    std::vector<float2> hOut(nb * nr * (nc / 2 + 1));
    checkCudaErrors(hipfftXtMemcpy(plan, (void*)hOut.data(), dX, HIPFFT_COPY_DEVICE_TO_HOST));
    checkCudaErrors(hipDeviceSynchronize());

    print3Dfloat2(hOut, nb, nr, (nc / 2 + 1));

    checkCudaErrors(hipfftXtFree(dX));
    checkCudaErrors(hipfftDestroy(plan));

    checkCudaErrors(hipDeviceReset());

    return 0;
}

//#include <assert.h>
//#include <hipfft/hipfft.h>
//#include <stdio.h>
//#include <stdlib.h>
//
// const size_t sigSize = 1 << 29;
// typedef hipfftComplex ctype;
// typedef hipfftReal rtype;
//
// int runMultiGPU() {
//
//    hipfftResult res;
//    rtype *devInData0, *devInData1;
//    ctype *devOutData0, *devOutData1;
//    hipfftHandle handle0, handle1;
//    hipStream_t stream0, stream1;
//
//    hipSetDevice(0);
//    res = hipfftPlan1d(&handle0, sigSize, HIPFFT_R2C, 1);
//    assert(res == HIPFFT_SUCCESS);
//    hipStreamCreate(&stream0);
//    res = hipfftSetStream(handle0, stream0);
//    assert(res == HIPFFT_SUCCESS);
//    hipMalloc(&devInData0, sizeof(rtype) * sigSize);
//    hipMalloc(&devOutData0, sizeof(ctype) * (sigSize * 2 + 1));
//
//    hipSetDevice(1);
//    res = hipfftPlan1d(&handle1, sigSize, HIPFFT_R2C, 1);
//    assert(res == HIPFFT_SUCCESS);
//    hipStreamCreate(&stream1);
//    res = hipfftSetStream(handle1, stream1);
//    assert(res == HIPFFT_SUCCESS);
//    hipMalloc(&devInData1, sizeof(rtype) * sigSize);
//    hipMalloc(&devOutData1, sizeof(ctype) * (sigSize * 2 + 1));
//
//    hipSetDevice(0);
//    res = hipfftExecR2C(handle0, devInData0, devOutData0);
//    assert(res == HIPFFT_SUCCESS);
//    hipDeviceSynchronize();
//
//    hipSetDevice(1);
//    res = hipfftExecR2C(handle1, devInData1, devOutData1);
//    assert(res == HIPFFT_SUCCESS);
//    hipDeviceSynchronize();
//
//    return 0;
//}

//#include "hipfft/hipfft.h"
//#include "hipfft/hipfftXt.h"
//#include "prettyprint.h"
//#include <cassert>
//#include <hip/hip_complex.h>
//#include <vector>
//
//#define CUDA_CHECK(x)  assert(x == hipSuccess)
//#define CUFFT_CHECK(x) assert(x == HIPFFT_SUCCESS)
//
// int runMultiGPU() {
//    static const int numGPUs = 2;
//    int gpus[numGPUs] = {0, 1};
//
//    int nr = 16;
//    int nc = 8;
//
//    // Fill with junk data
//    std::vector<hipFloatComplex> h_x(nr * nc);
//    for (int i = 0; i < nr * nc; ++i) {
//        h_x[i].x = static_cast<float>(i);
//    }
//
//    print3Dfloat2(h_x, 1, nr, nc);
//
//    hipfftHandle plan;
//    CUFFT_CHECK(hipfftCreate(&plan));
//    CUFFT_CHECK(hipfftXtSetGPUs(plan, numGPUs, gpus));
//
//    std::vector<size_t> workSizes(numGPUs);
//    int n[] = {nr};
//
//    //    // dimension of fft
//    //    int rank = 2;
//    //    Array of size rank, describing the size of each dimension, n[0] being the size of the
//    //    outermost and n[rank-1] innermost (contiguous) dimension of a transform.
//    //    int n[2] = {nr, nc};
//    //    // dist between batches
//    //    int iDist = nr * nc;
//    //    int oDist = nr * (nc / 2 + 1);
//    //    // input/output sizes with pitches ("unpitched")
//    //    int inEmbed[] = {nr, nc};
//    //    int onEmbed[] = {nr, nc / 2 + 1};
//    //    // stride between adjacent entries in row
//    //    int iStride = 1;
//    //    int oStride = 1;
//
//    CUFFT_CHECK(hipfftMakePlanMany(plan,
//                                  1, // rank
//                                  n, // n
//                                  n, // inembed
//                                  1, // istride
//                                  1, // idist
//                                  n, // onembed
//                                  1, // ostride
//                                  1, // odist
//                                  HIPFFT_C2C,
//                                  nc,
//                                  workSizes.data()));
//
//    hipLibXtDesc* d_x;
//    CUFFT_CHECK(hipfftXtMalloc(plan, &d_x, HIPFFT_XT_FORMAT_INPLACE));
//
//    CUFFT_CHECK(hipfftXtMemcpy(plan, d_x, (void*)h_x.data(), HIPFFT_COPY_HOST_TO_DEVICE));
//
//    CUFFT_CHECK(hipfftXtExecDescriptorC2C(plan, d_x, d_x, HIPFFT_FORWARD));
//
//    std::vector<float2> h_out(nr * nc);
//    CUFFT_CHECK(hipfftXtMemcpy(plan, (void*)h_out.data(), d_x, HIPFFT_COPY_DEVICE_TO_HOST));
//
//    print3Dfloat2(h_out, 1, nr, nc);
//
//    //    CUFFT_CHECK(hipfftXtFree(d_x));
//    //    CUFFT_CHECK(hipfftDestroy(plan));
//    //
//    //    CUDA_CHECK(hipDeviceReset());
//
//    return 0;
//}