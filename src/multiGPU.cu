#include "hip/hip_runtime.h"
//
// Created by Maksim Levental on 12/28/20.
//

#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <vector>

#include "hip/hip_runtime_api.h"
#include "multiGPU.cuh"
#include "stacktrace.h"
#include "util.h"

void multiplyCoefficient(float2* signal,
                         hipLibXtDesc* kernel,
                         int nGPUs,
                         int batchSize,
                         int imgHeight,
                         int imgWidth);

int runMultiGPU() {
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* - - - load image using OpenCV - - - */
    cv::Mat imgCv;
    CUDA_TIME([&]() {
        auto fp = "/home/max/dev_projects/cuda_blob/data/S_000_1752450056/"
                  "Tile_r1-c1_S_000_1752450056.tif";
        imgCv = imread(fp, cv::IMREAD_GRAYSCALE);
        if (imgCv.empty()) {
            std::cout << "Could not read the image: " << fp << std::endl;
            return 1;
        }
        imgCv.convertTo(imgCv, CV_32FC1);
    })
    printf("img load and convert to float time %.5fms\n", milliseconds);
    cv::resize(imgCv, imgCv, cv::Size(8192, 8192));
    int imgWidth = imgCv.cols;
    int imgHeight = imgCv.rows;
    int batchSize = 15;
    printf("width %d, height %d batch size %d\n", imgWidth, imgHeight, batchSize);
    assert(imgCv.channels() == 1);

    /* - - - Building the Kernel with 0-padding to the size of the image - - - */
    // The real-to-complex transform is implicitly a forward transform. For an in-place
    // real-to-complex transform where FFTW compatible output is desired, the input size must be
    // padded to ⌊N/2⌋ + 1 complex elements.
    // wtf?
    // something having to do with padding again???
    size_t totalSize = (size_t)batchSize * (size_t)imgHeight * ((size_t)imgWidth + 2);
    std::vector<float> kernelH(totalSize);
    std::fill(kernelH.begin(), kernelH.begin() + totalSize, 0);
    int y, x;
    size_t zzyyxx;
    CUDA_TIME([&]() {
        for (int k = 0; k < batchSize; k++) {
            auto radius = 1;
            auto kernel = gaussianKernel(2 * radius + 1);
            for (int i = ((imgHeight / 2) - radius); i <= ((imgHeight / 2) + radius); i++) {
                for (int j = ((imgWidth / 2) - radius); j <= ((imgWidth / 2) + radius); j++) {
                    y = i - ((imgHeight / 2) - radius);
                    x = j - ((imgWidth / 2) - radius);
                    zzyyxx = k * (imgHeight * imgWidth) + i * imgWidth + j;
                    kernelH[zzyyxx] = kernel[y][x];
                }
            }
        }
    })
    printf("kernel creation time %.5fms\n", milliseconds);

    ////////////////////// doesn't work
    //    print3Dfloat(reinterpret_cast<float*>(imgCv.data), 1, imgHeight, imgWidth);
    //    print3Dfloat(kernelH.data(), batchSize, imgHeight, imgWidth + 2);

    /* - - -  ffts - - - */
    // img fft
    hipfftHandle imgForwardPlan, imgInversePlan;
    float* imgH = (float*)(imgCv.isContinuous() ? imgCv.data : imgCv.clone().data);
    float* imgD;
    float2* imgFreqsD;

    CUDA_TIME(
        [&]() { checkCudaErrors(hipfftPlan2d(&imgForwardPlan, imgWidth, imgHeight, HIPFFT_R2C)); })
    printf("img forward plan time %.5fms\n", milliseconds);
    CUDA_TIME(
        [&]() { checkCudaErrors(hipfftPlan2d(&imgInversePlan, imgWidth, imgHeight, HIPFFT_C2R)); })
    printf("img inverse plan time %.5fms\n", milliseconds);
    checkCudaErrors(hipMalloc(&imgD, sizeof(float) * imgHeight * imgWidth));
    checkCudaErrors(hipMalloc(&imgFreqsD, sizeof(float2) * imgHeight * (imgWidth / 2 + 1)));

    CUDA_TIME([&]() {
        checkCudaErrors(
            hipMemcpy(imgD, imgH, sizeof(float) * imgHeight * imgWidth, hipMemcpyHostToDevice));
    })
    printf("img copy to device time %.5fms\n", milliseconds);
    CUDA_TIME([&]() { checkCudaErrors(hipfftExecR2C(imgForwardPlan, imgD, imgFreqsD)); })
    printf("img fft time %.5fms\n", milliseconds);

    // kernel fft
    static const int numGPUs = 2;
    int gpus[numGPUs] = {0, 1};

    hipfftHandle kernelForwardPlan, kernelInversePlan;
    checkCudaErrors(hipfftCreate(&kernelForwardPlan));
    checkCudaErrors(hipfftCreate(&kernelInversePlan));
    checkCudaErrors(hipfftXtSetGPUs(kernelForwardPlan, numGPUs, gpus));
    checkCudaErrors(hipfftXtSetGPUs(kernelInversePlan, numGPUs, gpus));

    // dimension of fft
    int rank = 2;
    int n[2] = {imgHeight, imgWidth};
    // input/output sizes with pitches ("unpitched")
    int inEmbed[] = {imgHeight, imgWidth};
    int onEmbed[] = {imgHeight, imgWidth / 2 + 1};
    // dist between batches
    int iDist = imgHeight * imgWidth;
    int oDist = imgHeight * (imgWidth / 2 + 1);
    // stride between adjacent entries in row
    int iStride = 1;
    int oStride = 1;

    size_t workSize[2];
    CUDA_TIME([&]() {
        hipfftMakePlanMany(kernelForwardPlan,
                          rank,
                          n,
                          inEmbed,
                          iStride,
                          iDist,
                          onEmbed,
                          oStride,
                          oDist,
                          HIPFFT_R2C,
                          batchSize,
                          workSize);
        hipfftMakePlanMany(kernelInversePlan,
                          rank,
                          n,
                          onEmbed,
                          oStride,
                          oDist,
                          inEmbed,
                          iStride,
                          iDist,
                          HIPFFT_C2R,
                          batchSize,
                          workSize);
    })
    printf("kernel plan time %.5fms\n", milliseconds);
    hipLibXtDesc* kernelFreqsDDesc;
    checkCudaErrors(hipfftXtMalloc(kernelForwardPlan, &kernelFreqsDDesc, HIPFFT_XT_FORMAT_INPLACE));
    CUDA_TIME([&]() {
        checkCudaErrors(hipfftXtMemcpy(
            kernelForwardPlan, kernelFreqsDDesc, kernelH.data(), HIPFFT_COPY_HOST_TO_DEVICE));
    })
    printf("kernel copy to device time %.5fms\n", milliseconds);
    CUDA_TIME([&]() {
        checkCudaErrors(
            hipfftXtExecDescriptorR2C(kernelForwardPlan, kernelFreqsDDesc, kernelFreqsDDesc));
    })
    printf("kernel fft time %.5fms\n", milliseconds);

    //    std::vector<float2> hOut(batchSize * imgHeight * (imgWidth / 2 + 1));
    //    checkCudaErrors(hipfftXtMemcpy(
    //        kernelForwardPlan, (void*)kernelFreqsD, kernelFreqsDDesc,
    //        HIPFFT_COPY_DEVICE_TO_DEVICE));
    //    checkCudaErrors(hipDeviceSynchronize());

    //    print3Dfloat2(hOut, batchSize, imgHeight, (imgWidth / 2 + 1));

    //    printf("\n\nValue of Library Descriptor\n");
    //    printf("Number of GPUs %d\n", kernelFreqsDDesc->descriptor->nGPUs);
    //    printf("Device id  %d %d\n",
    //           kernelFreqsDDesc->descriptor->GPUs[0],
    //           kernelFreqsDDesc->descriptor->GPUs[1]);
    //    printf("Data size on GPU %ld %ld\n",
    //           (long)(kernelFreqsDDesc->descriptor->size[0] / sizeof(hipfftComplex)),
    //           (long)(kernelFreqsDDesc->descriptor->size[1] / sizeof(hipfftComplex)));

    // Multiply the coefficients together and normalize the result
    CUDA_TIME([&]() {
        multiplyCoefficient(imgFreqsD, kernelFreqsDDesc, numGPUs, batchSize, imgHeight, imgWidth);
    })
    printf("filtering time %.5fms\n", milliseconds);

    CUDA_TIME([&]() {
        checkCudaErrors(
            hipfftXtExecDescriptorC2R(kernelInversePlan, kernelFreqsDDesc, kernelFreqsDDesc));
    })
    printf("filtered inverse fft time %.5fms\n", milliseconds);
    std::vector<float> filteredH(totalSize);
    CUDA_TIME([&]() {
        checkCudaErrors(hipfftXtMemcpy(
            kernelInversePlan, filteredH.data(), kernelFreqsDDesc, HIPFFT_COPY_DEVICE_TO_HOST));
    })
    printf("filtered copy to host time %.5fms\n", milliseconds);

    //    print3Dfloat(filteredH.data(), batchSize, imgHeight, imgWidth + 2);

    checkCudaErrors(hipfftXtFree(kernelFreqsDDesc));
    checkCudaErrors(hipfftDestroy(kernelForwardPlan));
    checkCudaErrors(hipDeviceReset());

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
//  Launch Kernel on multiple GPU
////////////////////////////////////////////////////////////////////////////////
void multiplyCoefficient(float2* signal,
                         hipLibXtDesc* kernel,
                         int nGPUs,
                         int batchSize,
                         int imgHeight,
                         int imgWidth) {
    auto numThreads = 32;
    dim3 dimBlock(numThreads, numThreads);
    int nBlocksW = (imgWidth / 2 + 1) / numThreads;
    if (((imgWidth / 2 + 1) % numThreads) != 0) nBlocksW++;
    int nBlocksH = imgHeight / numThreads;
    if ((imgHeight % numThreads) != 0) nBlocksH++;
    dim3 dimGrid(nBlocksW, nBlocksH, batchSize);

    int imgSize = sizeof(float2) * imgHeight * (imgWidth / 2 + 1);
    int origDevice;
    checkCudaErrors(hipGetDevice(&origDevice));
    int device;
    for (int i = 0; i < nGPUs; i++) {
        device = kernel->descriptor->GPUs[i];
        checkCudaErrors(hipSetDevice(device));
        float2* localSignal;
        if (device != origDevice) {
            checkCudaErrors(hipMalloc(&localSignal, imgSize));
            checkCudaErrors(hipMemcpyPeer(localSignal, device, signal, 0, imgSize));
        } else {
            localSignal = signal;
        }
        componentwiseMatrixMul1vsBatchfloat2<<<dimGrid, dimBlock>>>(
            localSignal,
            (float2*)kernel->descriptor->data[i],
            (float2*)kernel->descriptor->data[i],
            batchSize / nGPUs,
            imgHeight,
            (imgWidth / 2 + 1));
    }

    for (int i = 0; i < nGPUs; i++) {
        device = kernel->descriptor->GPUs[i];
        checkCudaErrors(hipSetDevice(device));
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("Kernel execution failed [ componentwiseMatrixMul1vsBatchfloat2 ]");
    }
    checkCudaErrors(hipSetDevice(origDevice));
}

//#include <assert.h>
//#include <hipfft/hipfft.h>
//#include <stdio.h>
//#include <stdlib.h>
//
// const size_t sigSize = 1 << 29;
// typedef hipfftComplex ctype;
// typedef hipfftReal rtype;
//
// int runMultiGPU() {
//
//    hipfftResult res;
//    rtype *devInData0, *devInData1;
//    ctype *devOutData0, *devOutData1;
//    hipfftHandle handle0, handle1;
//    hipStream_t stream0, stream1;
//
//    hipSetDevice(0);
//    res = hipfftPlan1d(&handle0, sigSize, HIPFFT_R2C, 1);
//    assert(res == HIPFFT_SUCCESS);
//    hipStreamCreate(&stream0);
//    res = hipfftSetStream(handle0, stream0);
//    assert(res == HIPFFT_SUCCESS);
//    hipMalloc(&devInData0, sizeof(rtype) * sigSize);
//    hipMalloc(&devOutData0, sizeof(ctype) * (sigSize * 2 + 1));
//
//    hipSetDevice(1);
//    res = hipfftPlan1d(&handle1, sigSize, HIPFFT_R2C, 1);
//    assert(res == HIPFFT_SUCCESS);
//    hipStreamCreate(&stream1);
//    res = hipfftSetStream(handle1, stream1);
//    assert(res == HIPFFT_SUCCESS);
//    hipMalloc(&devInData1, sizeof(rtype) * sigSize);
//    hipMalloc(&devOutData1, sizeof(ctype) * (sigSize * 2 + 1));
//
//    hipSetDevice(0);
//    res = hipfftExecR2C(handle0, devInData0, devOutData0);
//    assert(res == HIPFFT_SUCCESS);
//    hipDeviceSynchronize();
//
//    hipSetDevice(1);
//    res = hipfftExecR2C(handle1, devInData1, devOutData1);
//    assert(res == HIPFFT_SUCCESS);
//    hipDeviceSynchronize();
//
//    return 0;
//}
