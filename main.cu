#include <iostream>
#include <mpi.h>
#include "stacktrace.h"
#include "prettyprint.h"

template<typename T>
void print_array(const T(& a), uint N, std::ostream& o = std::cout) {
    o << "{";
    for (std::size_t i = 0; i < N - 1; ++i) {
        o << a[i] << ", ";
    }
    o << a[N - 1] << "}\n";
}

#define MPI_CHECK(call) \
    if((call) != MPI_SUCCESS) { \
        printf("MPI error calling \"%s\"\n", #call); \
        MPI_Abort(MPI_COMM_WORLD, -1); }

#define CHECKCUDAERRORS(err)                                                                       \
    do {                                                                                           \
        if (err != hipSuccess) {                                                                  \
            fprintf(                                                                               \
                stderr,                                                                            \
                "CHECKCUDAERRORS() API error = %04d \"%s\" from file <%s>, line %i.\n",            \
                err,                                                                               \
                hipGetErrorString(err),                                                           \
                __FILE__,                                                                          \
                __LINE__);                                                                         \
            fprintf(stderr, "%d\n", hipSuccess);                                                  \
            print_stacktrace();                                                                    \
            exit(-1);                                                                              \
        }                                                                                          \
    } while (0)

int main() {
    MPI_Init(nullptr, nullptr);
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    double* x_h, * y_h; // host data
    double* x_d, * y_d; // device data
    const int N = 100,
            nBytes = N * sizeof(double);

    x_h = new double[N];
    y_h = new double[N];

    // allocate memory on device
    for (int i = 0; i < N; i++) {
        x_h[i] = (i % 137) + 1;
    }

    CHECKCUDAERRORS(hipSetDevice(world_rank));
    // copy data:  host --> device
    if (world_rank == 0) {
        print_array(x_h, N);
        CHECKCUDAERRORS(hipMalloc((void**) &x_d, nBytes));
        CHECKCUDAERRORS(hipMemcpy(x_d, x_h, nBytes, hipMemcpyHostToDevice));
        MPI_CHECK(MPI_Send(x_d, N, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD));
        printf("successfully sent\n");
    }

    if (world_rank == 1) {
        CHECKCUDAERRORS(hipMalloc((void**) &y_d, nBytes));
        MPI_Status status;
        MPI_CHECK(MPI_Recv(y_d, N, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status));
        printf("successfully received\n");
        CHECKCUDAERRORS(hipMemcpy(y_h, y_d, nBytes, hipMemcpyDeviceToHost));
        print_array(y_h, N);
    }

    delete[] x_h;
    delete[] y_h;
    hipFree(x_d);
    hipFree(y_d);
    MPI_Finalize();
    return 0;
}